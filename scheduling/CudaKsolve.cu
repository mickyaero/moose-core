
#include <hip/hip_runtime.h>
/***
 *       Filename:  CudaKsolve.cu
 *
 *    Description:  CUDA version of Ksolve.
 *
 *        Version:  0.0.1
 *        Created:  2017-06-25
 *       Revision:  none
 *
 *         Author:  Micky Droch <mickydroch@gmail.com>
 *   Organization:  IIT Bombay
 *
 *        License:  GNU GPL3
 *
 *
 *        TODO: Needs to figure out a better location for this function.
 */

#ifdef USE_CUDA
#include <stdio.h>
#include "CudaKsolve.h"

#include "../basecode/header.h"
#include "../ksolve/VoxelPools.h"
#include "../ksolve/RateTerm.h"
#include "../ksolve/BoostSys.h"

inline void callMe( )
{
    printf( "Calling me\n" );
}

inline void cuda_ksolve( double* dy, double* y, const double currentTime, const double time, size_t n )
{

}

void voxelPoolToCudaOdeSystem( VoxelPoolsi& pool, CudaOdeSystem* pOde )
{
    // Get the Stoich first. It contains matrices we need.
    vector< double > yvec = pool.SInitVec( );

    pOde->dimension = yvec.size( );

    cout << "Volumne " << pool.getVolume( ) << endl;
    pOde->f = pool.varS( );

    BoostSys bs = pool.sys_;

#if 0
    vector<RateTerm*> vecRates = pool.getRateTerms( );
    for( auto r : vecRates )
        cout << r->getR1( ) << " " << r->getR2( ) << endl;
#endif

    //cout << "Total rate terms " << vecRates.size( ) << endl;
}


#endif
